#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <fstream>

#include "element_wise.cuh"
#include <kernels/registry.h>
#include <tensor.h>

namespace rwkv {
namespace cuda {

Tensor layer_norm_op(const Tensor &x, const Tensor &weight, const Tensor &bias);
Tensor group_norm_op(const Tensor &x, int num_groups, const Tensor &weight,
                     const Tensor &bias);

namespace {
// Equivalent Python code:
// ww = t_first + k
// p = torch.maximum(pp, ww)
// e1 = torch.exp(pp - p)
// e2 = torch.exp(ww - p)
// wkv = ((e1 * aa + e2 * v) / (e1 * bb + e2)).to(dtype=x.dtype)
// ww = t_decay + pp
// p = torch.maximum(ww, k)
// e1 = torch.exp(ww - p)
// e2 = torch.exp(k - p)
// t1 = e1 * aa + e2 * v
// t2 = e1 * bb + e2
// r = r * wkv
// return t1, t2, p, r
struct WkvForwardOne {
  const float *t_first;
  const float *k;
  const float *pp;
  const float *aa;
  const float *bb;
  const float *t_decay;
  const float *v;
  /* out */ float *t1;
  /* out */ float *t2;
  /* out */ float *p;
  /* in & out */ half *r;

  __device__ void operator()(int i) const {
    float ww = t_first[i] + k[i];
    float pp_ = pp[i];
    float p_ = (pp_ > ww) ? pp_ : ww;
    float e1 = expf(pp_ - p_);
    float e2 = expf(ww - p_);
    float aa_ = aa[i];
    float bb_ = bb[i];
    float v_ = v[i];
    r[i] = __hmul(r[i], __float2half(((e1 * aa_ + e2 * v_) / (e1 * bb_ + e2))));
    ww = t_decay[i] + pp_;
    float k_ = k[i];
    p_ = (ww > k_) ? ww : k_;
    e1 = expf(ww - p_);
    e2 = expf(k_ - p_);
    t1[i] = e1 * aa_ + e2 * v_;
    t2[i] = e1 * bb_ + e2;
    p[i] = p_;
  }
};

struct OneV5MulAdd {
  const int broadcasted_length;
  const float *t_first;
  const float *a;
  const float *s;
  const float *t_decay;
  /* out */ float *out_temp2;
  /* out */ float *decayed_s;

  __device__ void operator()(int i) const {
    float t_first_elem = t_first[i / broadcasted_length];
    float a_elem = a[i];
    float s_elem = s[i];
    float t_decay_elem = t_decay[i / broadcasted_length];
    out_temp2[i] = __fadd_rn(__fmul_rn(t_first_elem, a_elem), s_elem);
    decayed_s[i] = __fadd_rn(__fmul_rn(t_decay_elem, s_elem), a_elem);
  }
};

/*
   Equivalent Python code:
   kx = xx * k_mix + sx * (1 - k_mix)
   vx = xx * v_mix + sx * (1 - v_mix)
   rx = xx * r_mix + sx * (1 - r_mix)
*/

struct Mix {
  const half *xx;
  const half *sx;
  const half *k_mix;
  const half *v_mix;
  const half *r_mix;
  /* out */ half *kx;
  /* out */ half *vx;
  /* out */ half *rx;

  __device__ void operator()(int i) const {
    half xx_ = xx[i];
    half sx_ = sx[i];
    half k_mix_ = k_mix[i];
    half v_mix_ = v_mix[i];
    half r_mix_ = r_mix[i];
    kx[i] = __hadd(__hmul(xx_, k_mix_),
                   __hmul(sx_, __hsub(__float2half(1), k_mix_)));
    vx[i] = __hadd(__hmul(xx_, v_mix_),
                   __hmul(sx_, __hsub(__float2half(1), v_mix_)));
    rx[i] = __hadd(__hmul(xx_, r_mix_),
                   __hmul(sx_, __hsub(__float2half(1), r_mix_)));
  }
};

struct MixWithG {
  const half *xx;
  const half *sx;
  const half *k_mix;
  const half *v_mix;
  const half *r_mix;
  const half *g_mix;
  /* out */ half *kx;
  /* out */ half *vx;
  /* out */ half *rx;
  /* out */ half *gx;

  __device__ void operator()(int i) const {
    half xx_ = xx[i];
    half sx_ = sx[i];
    half k_mix_ = k_mix[i];
    half v_mix_ = v_mix[i];
    half r_mix_ = r_mix[i];
    half g_mix_ = g_mix[i];
    kx[i] = __hadd(__hmul(xx_, k_mix_),
                   __hmul(sx_, __hsub(__float2half(1), k_mix_)));
    vx[i] = __hadd(__hmul(xx_, v_mix_),
                   __hmul(sx_, __hsub(__float2half(1), v_mix_)));
    rx[i] = __hadd(__hmul(xx_, r_mix_),
                   __hmul(sx_, __hsub(__float2half(1), r_mix_)));
    gx[i] = __hadd(__hmul(xx_, g_mix_),
                   __hmul(sx_, __hsub(__float2half(1), g_mix_)));
  }
};

struct InplaceSiLU {
  half *x;
  __device__ void operator()(int i) const {
    half value = x[i];
    x[i] = __hdiv(value, __hadd(__float2half(1.0f),
                                __float2half(__expf(-__half2float(value)))));
  }
};

struct InplaceSigmoid {
  __device__ __forceinline__ void operator()(int i) const {
    ptr[i] = __float2half(1.0 / (1.0 + exp(-__half2float(ptr[i]))));
  }
  half *ptr;
};

struct InplaceAdd {
  __device__ __forceinline__ void operator()(int i) const {
    y[i] = __hadd(x[i], y[i]);
  }
  half *y;
  const half *x;
};

struct InplaceMulOther {
  __device__ __forceinline__ void operator()(int i) const {
    x[i] = __hmul(x[i], other[i]);
  }
  half *x;
  const half *other;
};

} // namespace

void gemm_cublas_tensor(const Tensor &a, const Tensor &b, Tensor &c);
Tensor cast_dtype(const Tensor &x, DType dtype);

static void save_tensor(const Tensor &_t, const std::string &name) {
  return;
  auto t = Copy(_t, Device::kCPU);
  std::ofstream f("/tmp/" + name, std::ios::binary | std::ios::out);
  if (t.dtype() == DType::kFloat16) {
    for (int i = 0; i < t.numel(); ++i) {
      f.write((char *)&t.data_ptr<float16>()[i], sizeof(float16));
    }
  } else if (t.dtype() == DType::kFloat32) {
    for (int i = 0; i < t.numel(); ++i) {
      f.write((char *)&t.data_ptr<float>()[i], sizeof(float));
    }
  } else {
    RV_UNIMPLEMENTED();
  }
}

Tensor _ATT(const Tensor &x, const Tensor &ln_w, const Tensor &ln_b,
            const Tensor &sx, const Tensor &k_mix, const Tensor &v_mix,
            const Tensor &r_mix, const Tensor &kw,
            /* imm */ Tensor &kx, const Tensor &vw, /* imm */ Tensor &vx,
            const Tensor &rw,
            /* imm */ Tensor &rx, const Tensor &ow, const Tensor &t_first,
            /* imm */ Tensor &k, const Tensor &pp, const Tensor &ww,
            const Tensor &aa, const Tensor &bb, const Tensor &t_decay,
            /* imm */ Tensor &v, /* in & out */ Tensor &r,
            /* out */ Tensor &x_plus_out, /* out */ Tensor &t1,
            /* out */ Tensor &t2, /* out */ Tensor &p) {
  static int i = 0;
  save_tensor(x, "x_att_fr" + std::to_string(i));
  Tensor xx = cuda::layer_norm_op(x, ln_w, ln_b);
  save_tensor(xx, "xx_att_fr" + std::to_string(i));
  element_wise(Mix{xx.data_ptr<half>(), sx.data_ptr<half>(),
                   k_mix.data_ptr<half>(), v_mix.data_ptr<half>(),
                   r_mix.data_ptr<half>(), kx.data_ptr<half>(),
                   vx.data_ptr<half>(), rx.data_ptr<half>()},
               x.numel());
  save_tensor(sx, "sx_att_fr" + std::to_string(i));
  save_tensor(k_mix, "k_mix_att_fr" + std::to_string(i));
  save_tensor(v_mix, "v_mix_att_fr" + std::to_string(i));
  save_tensor(r_mix, "r_mix_att_fr" + std::to_string(i));
  save_tensor(kx, "kx_att_fr" + std::to_string(i));
  save_tensor(vx, "vx_att_fr" + std::to_string(i));
  save_tensor(rx, "rx_att_fr" + std::to_string(i));

  gemm_cublas_tensor(kx, kw, k);
  save_tensor(kw, "kw_att_fr" + std::to_string(i));
  save_tensor(k, "k_att_fr" + std::to_string(i));
  gemm_cublas_tensor(vx, vw, v);
  save_tensor(vw, "vw_att_fr" + std::to_string(i));
  save_tensor(v, "v_att_fr" + std::to_string(i));
  gemm_cublas_tensor(rx, rw, r);
  save_tensor(rw, "rw_att_fr" + std::to_string(i));
  save_tensor(r, "r_att_fr_before_sigmoid" + std::to_string(i));
  element_wise(InplaceSigmoid{r.data_ptr<half>()}, r.numel());
  save_tensor(r, "r_att_fr" + std::to_string(i));

  element_wise(WkvForwardOne{t_first.data_ptr<float>(), k.data_ptr<float>(),
                             pp.data_ptr<float>(), aa.data_ptr<float>(),
                             bb.data_ptr<float>(), t_decay.data_ptr<float>(),
                             v.data_ptr<float>(), t1.data_ptr<float>(),
                             t2.data_ptr<float>(), p.data_ptr<float>(),
                             r.data_ptr<half>()},
               x.numel());
  save_tensor(t_first, "t_first_att_fr" + std::to_string(i));
  save_tensor(pp, "pp_att_fr" + std::to_string(i));
  save_tensor(aa, "aa_att_fr" + std::to_string(i));
  save_tensor(bb, "bb_att_fr" + std::to_string(i));
  save_tensor(t_decay, "t_decay_att_fr" + std::to_string(i));
  save_tensor(t1, "t1_att_fr" + std::to_string(i));
  save_tensor(t2, "t2_att_fr" + std::to_string(i));
  save_tensor(p, "p_att_fr" + std::to_string(i));
  save_tensor(r, "rwkv_att_fr" + std::to_string(i));

  gemm_cublas_tensor(r, ow, x_plus_out);
  save_tensor(ow, "ow_att_fr" + std::to_string(i));
  save_tensor(x_plus_out, "x_plus_out_att_fr_before_add" + std::to_string(i));
  element_wise(InplaceAdd{x_plus_out.data_ptr<half>(), x.data_ptr<half>()},
               x.numel());
  save_tensor(x_plus_out, "x_plus_out_att_fr" + std::to_string(i));
  i++;
  return xx;
}

Tensor _ATT_ONE_V5(const Tensor &x, const Tensor &s, const Tensor &ln_w,
                   const Tensor &ln_b, const Tensor &lx_w, const Tensor &lx_b,
                   const Tensor &sx, const Tensor &k_mix, const Tensor &v_mix,
                   const Tensor &r_mix, const Tensor &kw, Tensor &kx,
                   const Tensor &vw, Tensor &vx, const Tensor &rw, Tensor &rx,
                   const Tensor &ow, const Tensor &t_first, Tensor &k,
                   const Tensor &t_decay, Tensor &v, Tensor &r,
                   Tensor &decayed_s, Tensor &x_plus_out, Tensor &a,
                   Tensor &out_temp1, Tensor &out_temp2, LengthType H,
                   LengthType S) {
  Tensor xx = cuda::layer_norm_op(x, ln_w, ln_b);
  print_tensor(xx, "xx");

  element_wise(Mix{xx.data_ptr<half>(), sx.data_ptr<half>(),
                   k_mix.data_ptr<half>(), v_mix.data_ptr<half>(),
                   r_mix.data_ptr<half>(), kx.data_ptr<half>(),
                   vx.data_ptr<half>(), rx.data_ptr<half>()},
               x.numel());

  print_tensor(kx, "kx");
  print_tensor(vx, "vx");
  print_tensor(rx, "rx");
  gemm_cublas_tensor(kx, kw, k);
  gemm_cublas_tensor(vx, vw, v);
  gemm_cublas_tensor(rx, rw, r);

  r = r.view({H, 1, S});
  k = k.view({H, S, 1});
  v = v.view({H, 1, S});

  gemm_cublas_tensor(k, v, a);
  print_tensor(a, "a");

  element_wise(OneV5MulAdd{static_cast<int>(s.size(1) * s.size(2)),
                           t_first.data_ptr<float>(), a.data_ptr<float>(),
                           s.data_ptr<float>(), t_decay.data_ptr<float>(),
                           out_temp2.data_ptr<float>(),
                           decayed_s.data_ptr<float>()},
               s.numel());

  gemm_cublas_tensor(r, out_temp2, out_temp1);
  out_temp1 = out_temp1.flatten().unsqueeze(0);
  print_tensor(out_temp1, "out_temp1");

  Tensor out_temp3 = cuda::group_norm_op(out_temp1, H, lx_w, lx_b).flatten();
  print_tensor(out_temp3, "out_temp3");

  Tensor out_temp4 = cast_dtype(out_temp3, DType::kFloat16);
  gemm_cublas_tensor(out_temp4, ow, x_plus_out);
  element_wise(InplaceAdd{x_plus_out.data_ptr<half>(), x.data_ptr<half>()},
               x.numel());
  print_tensor(x_plus_out, "x_plus_out");

  return xx;
}

Tensor _ATT_ONE_V5_1(const Tensor &x, const Tensor &s, const Tensor &ln_w,
                     const Tensor &ln_b, const Tensor &lx_w, const Tensor &lx_b,
                     const Tensor &sx, const Tensor &k_mix, const Tensor &v_mix,
                     const Tensor &r_mix, const Tensor &g_mix, const Tensor &kw,
                     Tensor &kx, const Tensor &vw, Tensor &vx, const Tensor &rw,
                     Tensor &rx, const Tensor &gw, Tensor &gx, const Tensor &ow,
                     const Tensor &t_first, Tensor &k, const Tensor &t_decay,
                     Tensor &v, Tensor &r, Tensor &g, Tensor &decayed_s,
                     Tensor &x_plus_out, Tensor &a, Tensor &out_temp1,
                     Tensor &out_temp2, LengthType H, LengthType S) {
  Tensor xx = cuda::layer_norm_op(x, ln_w, ln_b);

  element_wise(MixWithG{xx.data_ptr<half>(), sx.data_ptr<half>(),
                        k_mix.data_ptr<half>(), v_mix.data_ptr<half>(),
                        r_mix.data_ptr<half>(), g_mix.data_ptr<half>(),
                        kx.data_ptr<half>(), vx.data_ptr<half>(),
                        rx.data_ptr<half>(), gx.data_ptr<half>()},
               x.numel());

  gemm_cublas_tensor(kx, kw, k);
  gemm_cublas_tensor(vx, vw, v);
  gemm_cublas_tensor(rx, rw, r);
  gemm_cublas_tensor(gx, gw, g);
  element_wise(InplaceSiLU{g.data_ptr<half>()}, g.numel());

  r = r.view({H, 1, S});
  k = k.view({H, S, 1});
  v = v.view({H, 1, S});

  gemm_cublas_tensor(k, v, a);

  // print_n(a, "a", a.numel() - 30, 30);
  // print_n(s, "s", s.numel() - 30, 30);
  // print_n(t_first, "t_first", t_first.numel() - 30, 30);

  element_wise(OneV5MulAdd{static_cast<int>(s.size(2)),
                           t_first.data_ptr<float>(), a.data_ptr<float>(),
                           s.data_ptr<float>(), t_decay.data_ptr<float>(),
                           out_temp2.data_ptr<float>(),
                           decayed_s.data_ptr<float>()},
               s.numel());

  // print_n(r, "r", r.numel() - 30, 30);
  // print_n(out_temp2, "out_temp2", out_temp2.numel() - 30, 30);
  gemm_cublas_tensor(r, out_temp2, out_temp1);
  out_temp1 = out_temp1.flatten().unsqueeze(0);

  // print_n(out_temp1, "out_temp1", out_temp1.numel() - 30, 30);

  Tensor out_temp3 = cuda::group_norm_op(out_temp1, H, lx_w, lx_b).flatten();

  // print_n(g, "g", g.numel() - 30, 30);
  // print_n(out_temp3, "out_temp3", out_temp3.numel() - 30, 30);

  Tensor out_temp4 = cast_dtype(out_temp3, DType::kFloat16);
  element_wise(InplaceMulOther{out_temp4.data_ptr<half>(), g.data_ptr<half>()},
               out_temp4.numel());
  // print_n(out_temp4, "out_temp4", out_temp4.numel() - 30, 30);
  gemm_cublas_tensor(out_temp4, ow, x_plus_out);
  // print_n(x_plus_out, "x_plus_out", x_plus_out.numel() - 30, 30);
  // print_n(ow, "ow", ow.numel() - 30, 30);
  element_wise(InplaceAdd{x_plus_out.data_ptr<half>(), x.data_ptr<half>()},
               x.numel());

  // print_n(x_plus_out, "x_plus_out", x_plus_out.numel() - 30, 30);
  // exit(0);

  return xx;
}

inline std::tuple<Tensor, Tensor, Tensor, Tensor, Tensor>
att(const Tensor &x, const Tensor &sx, const Tensor &aa, const Tensor &bb,
    const Tensor &pp, const Tensor &ln_w, const Tensor &ln_b,
    const Tensor &k_mix, const Tensor &v_mix, const Tensor &r_mix,
    const Tensor &t_decay, const Tensor &t_first, const Tensor &kw,
    const Tensor &vw, const Tensor &rw, const Tensor &ow) {

  auto kx = Tensor::Empty(x.sizes(), x.dtype(), x.device());
  auto vx = Tensor::Empty(x.sizes(), x.dtype(), x.device());
  auto rx = Tensor::Empty(x.sizes(), x.dtype(), x.device());
  auto k = Tensor::Empty({kw.size(0)}, DType::kFloat32, x.device());
  auto v = Tensor::Empty({vw.size(0)}, DType::kFloat32, x.device());
  auto r = Tensor::Empty({rw.size(0)}, DType::kFloat16, x.device());
  auto x_plus_out = Tensor::Empty(x.sizes(), x.dtype(), x.device());
  auto t1 = Tensor::Empty(x.sizes(), DType::kFloat32, x.device());
  auto t2 = Tensor::Empty(x.sizes(), DType::kFloat32, x.device());
  auto p = Tensor::Empty(x.sizes(), DType::kFloat32, x.device());

  Tensor xx =
      _ATT(x, ln_w, ln_b, sx, k_mix, v_mix, r_mix, kw, kx, vw, vx, rw, rx, ow,
           t_first, k, pp, vw, aa, bb, t_decay, v, r, x_plus_out, t1, t2, p);
  return std::make_tuple(x_plus_out, xx, t1, t2, p);
}

// The v5 version implementation of att
inline std::tuple<Tensor, Tensor, Tensor>
att_one_v5(const Tensor &x, const Tensor &sx, const Tensor &s,
           const Tensor &ln_w, const Tensor &ln_b, const Tensor &lx_w,
           const Tensor &lx_b, const Tensor &k_mix, const Tensor &v_mix,
           const Tensor &r_mix, const Tensor &t_decay, const Tensor &t_first,
           const Tensor &kw, const Tensor &vw, const Tensor &rw,
           const Tensor &ow) {

  auto kx = Tensor::Empty(x.sizes(), x.dtype(), x.device());
  auto vx = Tensor::Empty(x.sizes(), x.dtype(), x.device());
  auto rx = Tensor::Empty(x.sizes(), x.dtype(), x.device());
  auto k = Tensor::Empty({kw.size(0)}, DType::kFloat32, x.device());
  auto v = Tensor::Empty({vw.size(0)}, DType::kFloat32, x.device());
  auto r = Tensor::Empty({rw.size(0)}, DType::kFloat32, x.device());
  auto x_plus_out = Tensor::Empty(x.sizes(), x.dtype(), x.device());

  auto H = t_decay.size(0);
  auto S = x.size(x.shape().size() - 1) / H;

  auto decayed_s = Tensor::Empty(s.sizes(), DType::kFloat32, s.device());
  auto a = Tensor::Empty(s.sizes(), DType::kFloat32, s.device());
  auto out_temp1 =
      Tensor::Empty({s.size(0), 1, s.size(2)}, DType::kFloat32, s.device());
  auto out_temp2 = Tensor::Empty(s.sizes(), DType::kFloat32, s.device());

  Tensor xx = _ATT_ONE_V5(x, s, ln_w, ln_b, lx_w, lx_b, sx, k_mix, v_mix, r_mix,
                          kw, kx, vw, vx, rw, rx, ow, t_first, k, t_decay, v, r,
                          decayed_s, x_plus_out, a, out_temp1, out_temp2, H, S);
  return std::make_tuple(x_plus_out, xx, decayed_s);
}

std::tuple<Tensor, Tensor, Tensor>
att_one_v5_1(const Tensor &x, const Tensor &sx, const Tensor &s,
             const Tensor &ln_w, const Tensor &ln_b, const Tensor &lx_w,
             const Tensor &lx_b, const Tensor &k_mix, const Tensor &v_mix,
             const Tensor &r_mix, const Tensor &g_mix, const Tensor &t_decay,
             const Tensor &t_first, const Tensor &kw, const Tensor &vw,
             const Tensor &rw, const Tensor &gw, const Tensor &ow) {

  auto kx = Tensor::Empty(x.sizes(), x.dtype(), x.device());
  auto vx = Tensor::Empty(x.sizes(), x.dtype(), x.device());
  auto rx = Tensor::Empty(x.sizes(), x.dtype(), x.device());
  auto gx = Tensor::Empty(x.sizes(), x.dtype(), x.device());
  auto k = Tensor::Empty({kw.size(0)}, DType::kFloat32, x.device());
  auto v = Tensor::Empty({vw.size(0)}, DType::kFloat32, x.device());
  auto r = Tensor::Empty({rw.size(0)}, DType::kFloat32, x.device());
  auto g = Tensor::Empty({gw.size(0)}, gx.dtype(), x.device());
  auto x_plus_out = Tensor::Empty(x.sizes(), x.dtype(), x.device());

  auto H = t_decay.size(0);
  auto S = x.size(x.shape().size() - 1) / H;

  auto decayed_s = Tensor::Empty(s.sizes(), DType::kFloat32, s.device());
  auto a = Tensor::Empty(s.sizes(), DType::kFloat32, s.device());
  auto out_temp1 =
      Tensor::Empty({s.size(0), 1, s.size(2)}, DType::kFloat32, s.device());
  auto out_temp2 = Tensor::Empty(s.sizes(), DType::kFloat32, s.device());

  Tensor xx = _ATT_ONE_V5_1(x, s, ln_w, ln_b, lx_w, lx_b, sx, k_mix, v_mix,
                            r_mix, g_mix, kw, kx, vw, vx, rw, rx, gw, gx, ow,
                            t_first, k, t_decay, v, r, g, decayed_s, x_plus_out,
                            a, out_temp1, out_temp2, H, S);
  return std::make_tuple(x_plus_out, xx, decayed_s);
}

KernelRegister att_reg("att", Device::kCUDA, att);
KernelRegister att_one_v5_reg("att_one_v5", Device::kCUDA, att_one_v5);
KernelRegister att_one_v5_1_reg("att_one_v5_1", Device::kCUDA, att_one_v5_1);

} // namespace cuda
} // namespace rwkv
